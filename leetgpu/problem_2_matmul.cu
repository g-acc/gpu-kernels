#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= M || col >= K) {
        return;
    }

    float ans = 0.0f;
    for (int i = 0; i < N; i++) {
        ans += A[row * N + i] * B[i * K + col];
    }
    C[row * K + col] = ans;
}

extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

